#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math.h>

// useful definitions
#define NUM_BLOCKS 8
#define NUM_THREADS_PER_BLOCK 16
#define REAL_SIZE (N*N)
#define COMPLEX_SIZE (N*(N/2+1))
#define PI 3.14159265358979323846
#define L 200.0
#define DIFF_COEFF ((2.0 * M_PI / L) * (2.0 * M_PI / L))

// error handler
static void HANDLEERROR( hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file,line);
        exit(1);
    }
}
#define HandleError(err) (HANDLEERROR(err, __FILE__, __LINE__))

// advances 
__global__ void runge_kutta_advance(double* dev_u, double* dev_v, double* dev_d2u, double* dev_d2v, int step) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= N || j >= N) return;

    int idx = i * N + j;

    double old_u = dev_u[idx];
    double old_v = dev_v[idx];

    dev_u[idx] = old_u + dt / step * dev_d2u[idx];
    dev_v[idx] = old_v + dt / step * dev_d2v[idx];

    __syncthreads();
}

// helper function to rescale matrix after inverse fft
__global__ void rescale(double* dev_matrix, double factor, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalSize = rows * cols;
    if (idx < totalSize) {
        dev_matrix[idx] *= factor;
    }
}

// function to swap pointers for runge-kutta step
inline void swap(double*& A, double*& B) {
    double* temp = A;
    A = B;
    B = temp;
}

// consider computing only u instead of both u and v
__global__ void compute_derivative(hipfftDoubleComplex* dev_au, hipfftDoubleComplex* dev_av, double D_u, double D_v) {
    // TODO
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= N || j >= N/2 + 1) return;

    int kx = (i <= N/2) ? i : i - N;
    int ky = j;

    double k_s = DIFF_COEFF * (kx * kx + ky * ky);

    int idx = i * (N/2 + 1) + j;

    hipfftDoubleComplex old_au = dev_au[idx];
    hipfftDoubleComplex old_av = dev_av[idx];

    dev_au[idx].x = -D_u * k_s * old_au.x;
    dev_au[idx].y = -D_u * k_s * old_au.y;

    dev_av[idx].x = -D_v * k_s * old_av.x;
    dev_av[idx].y = -D_v * k_s * old_av.y;

    __syncthreads(); // probably dont need this
}

// void fft(hipfftHandle plan_r2c, hipfftHandle* plan_c2r, double* dev_u, hipfftDoubleComplex* dev_au, double* dev_v, hipfftDoubleComplex* dev_av) {
//     hipfftExecD2Z(plan_r2c, dev_u, dev_au);
//     hipfftExecD2Z(plan_r2c, dev_v, dev_av);
// }

// main loop
int main(int argc, char* argv[]) {
    if (argc < 9) {
        printf("not enough arguments!\n");
    }

    // parse arguments
    const int N = atoi(argv[1]);
    const int D_u = atoi(argv[2]);
    const int D_v = atoi(argv[3]);
    const double a = atof(argv[4]);
    const double b = atof(argv[5]);
    const double c = atof(argv[6]);
    const double eps = atof(argv[7]);
    const int K = atoi(argv[8]);

    // parse seed
    long seed;
    if (argc == 9) {
        seed = atol(argv[9]);
    } else {
        seed = 42;
    }

    printf("N: %d\n", N);
    printf("D_u: %d\n", D_u);
    printf("D_v: %d\n", D_v);
    printf("a: %.2f\n", a);
    printf("b: %.2f\n", b);
    printf("c: %.2f\n", c);
    printf("eps: %.2f\n", eps);
    printf("K: %d\n", K);
    printf("seed: %ld\n", seed);

    // declare 
    __constant__ int dev_N;
    __constant__ double dev_dt;
    __constant__ int dev_Du;
    __constant__ int dev_Dv;

    hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_dt), &dt, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_Du), &D_u, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_Dv), &D_v, sizeof(int));

    // set the seed
    srand48(seed);
    double omega;

    // time steps
    int T = 100; // terminal time
    double dt = (double)T / (double)K;

    // initialize matrices
    double* u = (double*)malloc(REAL_SIZE*sizeof(double));
    double* v = (double*)malloc(REAL_SIZE*sizeof(double));

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            omega = drand48();
            u[i*N + j] = (a + c)/b - 4.5*omega;
            v[i*N + j] = (a + c)*(a + c)/(c * b)/(c * b);
        }
    }

    // initialize matrices on device
    double *dev_u, *dev_v, *dev_d2u, *dev_d2v;
    hipMalloc((void**)&dev_u, sizeof(double) * REAL_SIZE);
    hipMalloc((void**)&dev_v, sizeof(double) * REAL_SIZE);

    hipMalloc((void**)&dev_d2u, sizeof(double) * REAL_SIZE);
    hipMalloc((void**)&dev_d2v, sizeof(double) * REAL_SIZE);

    // Assuming you have host arrays `u` and `v` allocated and initialized
    hipMemcpy(dev_u, u, sizeof(double) * REAL_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_v, v, sizeof(double) * REAL_SIZE, hipMemcpyHostToDevice);

    hipfftDoubleComplex *dev_au, *dev_av;
    hipMalloc((void**)&dev_au, sizeof(hipfftDoubleComplex) * COMPLEX_SIZE);
    hipMalloc((void**)&dev_av, sizeof(hipfftDoubleComplex) * COMPLEX_SIZE);

    // create fft plans
    hipfftHandle plan_r2c, plan_c2r;
    hipfftPlan2d(&plan_r2c, N, N, HIPFFT_R2C);
    hipfftPlan2d(&plan_c2r, N, N, HIPFFT_C2R);
    
    // cufft does not normalize so make sure to divide by N*N after the inverse

    dim3 blockDim(16, 16);
    dim3 gridDim(((N/2 + 1) + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    // time step loop
    for (int t = 0; t < K; t++) {
        // forward fft (t1)
        hipfftExecD2Z(plan_r2c, dev_u, dev_au);
        hipfftExecD2Z(plan_r2c, dev_v, dev_av);
    
        // compute derivative
        compute_derivative<<<blockDim, gridDim>>>(dev_au, dev_av, N, D_u, D_v);
        
        // backward fft (t1)
        hipfftExecZ2D(plan_c2r, dev_d2u, dev_au);
        hipfftExecZ2D(plan_c2r, dev_d2v, dev_av);

        // runge-kutta time step (t1)

        // swap pointers

        // forward fft (t2)
        // backward fft (t2)
        // runge-kutta time step (t2)
        // swap pointers

        // forward fft (t3)
        // backward fft (t3)
        // runge-kutta time step (t3)
        // swap pointers

        // forward fft (t4)
        // backward fft (t4)
        // runge-kutta time step (t4)
        // swap pointers
        continue;
    }

    hipfftDestroy(plan_r2c); hipfftDestroy(plan_c2r);
    hipFree(dev_u); hipFree(dev_v); hipFree(dev_au); hipFree(dev_av);
    return 0;
}