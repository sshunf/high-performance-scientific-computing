#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#define NUM_BLOCKS 8;
#define NUM_THREADS_PER_BLOCK 16;
#define REAL_SIZE = N*N;
#define COMPLEX_SIZE = N*(N/2+1);

// error handler
static void HANDLEERROR( hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file,line);
        exit(1);
    }
}
#define HandleError(err) (HANDLEERROR(err, __FILE__, __LINE__))

__global__ void runge_kutta_advance(double* dev_u, double* dev_v, int step) {
    // TODO
    continue;
}

// helper function to rescale matrix after inverse fft
__global__ void rescale(double* dev_matrix, double* factor, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            dev_matrix[i * cols + j] *= scalar;
        }
    }
}

// function to swap pointers for runge-kutta step
inline void swap(double*& A, double*& B) {
    double* temp = A;
    A = B;
    B = temp;
}

int main(int argc, char* argv[]) {
    if (argc < 9) {
        printf("not enough arguments!\n");
    }

    // parse arguments
    int N = atoi(argv[1]);
    int D_u = atoi(argv[2]);
    int D_v = atoi(argv[3]);
    double a = atof(argv[4]);
    double b = atof(argv[5]);
    double c = atof(argv[6]);
    double eps = atof(argv[7]);
    int K = atoi(argv[8]);

    // parse seed
    long seed;
    if (argc == 9) {
        seed = atol(argv[9]);
    } else {
        seed = 42;
    }

    printf("N: %d\n", N);
    printf("D_u: %d\n", D_u);
    printf("D_v: %d\n", D_v);
    printf("a: %.2f\n", a);
    printf("b: %.2f\n", b);
    printf("c: %.2f\n", c);
    printf("eps: %.2f\n", eps);
    printf("K: %d\n", K);
    printf("seed: %ld\n", seed);

    // set the seed
    srand48(seed);
    double omega;
    printf("omega: %.2f\n", omega);

    // time steps
    int T = 100; // terminal time
    double dt = (double)T / (double)K;

    // initialize matrices
    double* u = (double*)malloc(REAL_SIZE*sizeof(double));
    double* v = (double*)malloc(REAL_SIZE*sizeof(double));

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            omega = drand48();
            u[i*cols + j] = (a + c)/b - 4.5*omega;
            v[i*cols + j] = (a + c)*(a + c)/(c * b)/(c * b);
        }
    }

    // initialize matrices on device
    double* dev_u, dev_v;
    hipMalloc(&dev_u, sizeof(double) * REAL_SIZE);
    hipMalloc(&dev_v, sizeof(double) * REAL_SIZE);

    hipMemcpy(dev_u, sizeof(double) * REAL_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_v, sizeof(double) * REAL_SIZE, hipMemcpyHostToDevice);

    hipfftDoubleComplex* dev_cu, dev_cv;
    hipMalloc((void**)&dev_cu, sizeof(hipfftDoubleComplex) * COMPLEX_SIZE);
    hipMalloc((void**)&dev_cv, sizeof(hipfftDoubleComplex) * COMPLEX_SIZE);

    // create fft plans
    hipfftHandle plan_r2c, plan_c2r;
    hipfftPlan2d(&plan_r2c, N, N, HIPFFT_R2C);
    hipfftPlan2d(&plan_c2r, N, N, HIPFFT_C2R);
    
    // cufft does not normalize so make sure to divide by N*N after the inverse

    // time step loop
    for (int t = 0; t < K; t++) {
        // forward fft (t1)
        // backward fft (t1)
        // runge-kutta time step (t1)
        // swap pointers

        // forward fft (t2)
        // backward fft (t2)
        // runge-kutta time step (t2)
        // swap pointers

        // forward fft (t3)
        // backward fft (t3)
        // runge-kutta time step (t3)
        // swap pointers

        // forward fft (t4)
        // backward fft (t4)
        // runge-kutta time step (t4)
        // swap pointers
    }

    hipfftDestroy(plan_r2c); hipfftDestroy(plan_c2r);
    hipFree(dev_u); hipFree(dev_v); hipFree(dev_cu); hipFree(dev_cv);
    return 0;
}